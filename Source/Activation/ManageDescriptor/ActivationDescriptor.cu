#include "ActivationDescriptor.h"
#include "Utilities/ErrorHandling/HandleUnsuccessfulCuDNNCall.h"

#include <hipDNN.h>

using Utilities::ErrorHandling::HandleUnsuccessfulCuDNNCall;

namespace Activation
{
namespace ManageDescriptor
{

ActivationDescriptor::ActivationDescriptor():
  descriptor_{}
{
  HandleUnsuccessfulCuDNNCall create_descriptor {
    "Failed to create Activation descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnCreateActivationDescriptor
  // 3.2.6. hipdnnCreateActivationDescriptor(). This function create an
  // activation descriptor object by allocating memory needed to hold its opaque
  // structure.
  create_descriptor(hipdnnCreateActivationDescriptor(&descriptor_));
}

ActivationDescriptor::~ActivationDescriptor()
{
  HandleUnsuccessfulCuDNNCall destroy_descriptor {
    "Failed to destroy Activation descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnDestroyTensorDescriptor
  destroy_descriptor(hipdnnDestroyActivationDescriptor(descriptor_));
}

} // namespace ManageDescriptor
} // namespace Activation