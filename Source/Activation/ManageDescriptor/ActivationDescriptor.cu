#include "ActivationDescriptor.h"
#include "Utilities/ErrorHandling/HandleUnsuccessfulCuDNNCall.h"

#include <hipDNN.h>

using Utilities::ErrorHandling::HandleUnsuccessfulCuDNNCall;

namespace Activation
{
namespace ManageDescriptor
{

ActivationDescriptor::ActivationDescriptor():
  descriptor_{}
{
  HandleUnsuccessfulCuDNNCall create_descriptor {
    "Failed to create Tensor descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnCreateActivationDescriptor
  // 3.2.6. hipdnnCreateActivationDescriptor(). This function create an
  // activation descriptor object by allocating memory needed to hold its opaque
  // structure.
  create_descriptor(hipdnnCreateActivationDescriptor(&descriptor_));
}

ActivationDescriptor::~ActivationDescriptor()
{
  HandleUnsuccessfulCuDNNCall destroy_descriptor {
    "Failed to destroy descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnDestroyTensorDescriptor
  destroy_descriptor(hipdnnDestroyActivationDescriptor(descriptor_));
}

} // namespace ManageDescriptor
} // namespace Activation