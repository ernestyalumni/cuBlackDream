#include "HandleUnsuccessfulCuDNNCall.h"

#include <hipDNN.h>
#include <iostream> // std::cerr
#include <string>

using std::cerr;

namespace Utilities
{
namespace ErrorHandling
{

HandleUnsuccessfulCuDNNCall::HandleUnsuccessfulCuDNNCall(
  const std::string& error_message
  ):
  error_message_{error_message},
  status_{HIPDNN_STATUS_SUCCESS}
{}

void HandleUnsuccessfulCuDNNCall::operator()(
  const hipdnnStatus_t cuDNN_status)
{
  status_ = cuDNN_status;

  if (!is_success())
  {
    cerr << error_message_ << " (error code " <<
      // https://docs.nvidia.com/deeplearning/cudnn/pdf/cuDNN-API.pdf
      // 3.2.45 cudnn_ops_infer.so Library
      // This function converts the cuDNN status code to a NULL terminated
      // (ASCIIZ) static string.
      hipdnnGetErrorString(status_) << ")!\n";
  }
}

} // namespace ErrorHandling
} // namespace Utilities