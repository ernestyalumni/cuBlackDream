#include "CuDNNLibraryHandle.h"
#include "Utilities/ErrorHandling/HandleUnsuccessfulCuDNNCall.h"

#include <hipDNN.h>

using Utilities::ErrorHandling::HandleUnsuccessfulCuDNNCall;

namespace DeepNeuralNetwork
{

CuDNNLibraryHandle::CuDNNLibraryHandle():
  handle_{}
{
  HandleUnsuccessfulCuDNNCall create_handle {"Failed to create handle"};
  create_handle(hipdnnCreate(&handle_));
}

CuDNNLibraryHandle::~CuDNNLibraryHandle()
{
  HandleUnsuccessfulCuDNNCall destroy_handle {"Failed to destroy handle"};

  destroy_handle(hipdnnDestroy(handle_));
}

} // namespace DeepNeuralNetwork