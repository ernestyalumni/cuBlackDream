#include "TensorDescriptor.h"
#include "Utilities/ErrorHandling/HandleUnsuccessfulCuDNNCall.h"

#include <hipDNN.h>

using Utilities::ErrorHandling::HandleUnsuccessfulCuDNNCall;

namespace Tensors
{
namespace ManageDescriptor
{

TensorDescriptor::TensorDescriptor():
  descriptor_{}
{
  HandleUnsuccessfulCuDNNCall create_descriptor {
    "Failed to create Tensor descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnCreateTensorDescriptor
  // 3.2.16. hipdnnCreateTensorDescriptor(). This function create a generic
  // tensor descriptor object by allocating memory needed to hold its opaque
  // structure. Data is initialized to all zeros.
  create_descriptor(hipdnnCreateTensorDescriptor(&descriptor_));
}

TensorDescriptor::~TensorDescriptor()
{
  HandleUnsuccessfulCuDNNCall destroy_descriptor {
    "Failed to destroy descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnDestroyTensorDescriptor
  destroy_descriptor(hipdnnDestroyTensorDescriptor(descriptor_));
}

} // namespace ManageDescriptor
} // namespace Tensors