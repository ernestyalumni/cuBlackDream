#include "Parameters.h"

#include <cstddef>
#include <hipDNN.h>
#include <stdexcept>

using std::runtime_error;
using std::size_t;

namespace RecurrentNeuralNetwork
{

Parameters::Parameters(
  hipdnnRNNAlgo_t algorithm,
  hipdnnRNNMode_t cell_mode,
  hipdnnRNNBiasMode_t bias_mode,
  hipdnnDirectionMode_t direction_mode,
  hipdnnRNNInputMode_t input_mode,
  hipdnnDataType_t data_type,
  hipdnnDataType_t math_precision,
  hipdnnMathType_t math_operation_type,
  const size_t input_size,
  const size_t hidden_size,
  const size_t projection_size,
  const size_t number_of_layers,
  const size_t maximum_sequence_length,
  const size_t batch_size,
  const uint32_t auxiliary_flags
  ):
  algo_{algorithm},
  cell_mode_{cell_mode},
  bias_mode_{bias_mode},
  direction_mode_{direction_mode},
  input_mode_{input_mode},
  data_type_{data_type},
  math_precision_{math_precision},
  math_type_{math_operation_type},
  input_size_{static_cast<int32_t>(input_size)},
  hidden_size_{static_cast<int32_t>(hidden_size)},
  projection_size_{static_cast<int32_t>(projection_size)},
  number_of_layers_{static_cast<int32_t>(number_of_layers)},
  maximum_sequence_length_{static_cast<int>(maximum_sequence_length)},
  batch_size_{static_cast<int>(batch_size)},
  auxiliary_flags_{auxiliary_flags}
{
  if (data_type_ == HIPDNN_DATA_HALF)
  {
    if ((math_type_ != HIPDNN_DEFAULT_MATH) &&
      (math_type_ != HIPDNN_TENSOR_OP_MATH))
    {
      throw runtime_error(
        "Since data type is HIPDNN_DATA_HALF, math type is default or tensor op only."
      );
    }
  }

  if (input_mode_ == HIPDNN_SKIP_INPUT)
  {
    if (input_size_ != hidden_size_)
    {
      throw runtime_error(
        "Since input mode is HIPDNN_SKIP_INPUT, input_size = hidden_size");
    }
  }

  if (projection_size > hidden_size)
  {
    throw runtime_error(
      "[ERROR] Inconsistent parameter: projSize is larger than hiddenSize!");
  }
}

DefaultParameters::DefaultParameters():
  Parameters{
    HIPDNN_RNN_ALGO_STANDARD,
    HIPDNN_RNN_RELU,
    HIPDNN_RNN_WITH_BIAS,
    HIPDNN_UNIDIRECTIONAL,
    HIPDNN_LINEAR_INPUT,
    HIPDNN_DATA_FLOAT,
    HIPDNN_DATA_FLOAT,
    HIPDNN_DEFAULT_MATH,
    512,
    512,
    512,
    2,
    20,
    64,
    CUDNN_RNN_PADDED_IO_DISABLED}
{}

} // namespace RecurrentNeuralNetwork
