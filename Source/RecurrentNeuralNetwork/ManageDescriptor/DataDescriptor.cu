#include "DataDescriptor.h"
#include "Utilities/ErrorHandling/HandleUnsuccessfulCuDNNCall.h"

#include <hipDNN.h>

using Utilities::ErrorHandling::HandleUnsuccessfulCuDNNCall;

namespace RecurrentNeuralNetwork
{
namespace ManageDescriptor
{

DataDescriptor::DataDescriptor():
  descriptor_{}
{
  HandleUnsuccessfulCuDNNCall create_descriptor {
    "Failed to create RNN Data descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnCreateRNNDescriptor
  // 7.2.6. hipdnnCreateRNNDescriptor(). This function create a generic RNN
  // descriptor object by allocating memory needed to hold its opaque structure.
  create_descriptor(cudnnCreateRNNDataDescriptor(&descriptor_));
}

DataDescriptor::~DataDescriptor()
{
  HandleUnsuccessfulCuDNNCall destroy_descriptor {
    "Failed to destroy RNN Data descriptor"};

  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnDestroyRNNDescriptor
  destroy_descriptor(cudnnDestroyRNNDataDescriptor(descriptor_));
}

} // namespace ManageDescriptor
} // namespace RecurrentNeuralNetwork