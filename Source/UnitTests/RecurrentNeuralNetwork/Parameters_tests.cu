#include "RecurrentNeuralNetwork/Parameters.h"
#include "gtest/gtest.h"

#include <hipDNN.h>

using RecurrentNeuralNetwork::DefaultParameters;

namespace GoogleUnitTests
{
namespace RecurrentNeuralNetwork
{

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(DefaultParametersTests, Constructs)
{
  DefaultParameters parameters {};

  // HIPDNN_RNN_ALGO_STANDARD
  EXPECT_EQ(static_cast<int>(parameters.algo_), 0);
  // HIPDNN_RNN_RELU
  EXPECT_EQ(static_cast<int>(parameters.cell_mode_), 0);
  // HIPDNN_RNN_WITH_BIAS
  EXPECT_EQ(static_cast<int>(parameters.bias_mode_), 2);
  EXPECT_EQ(parameters.bias_mode_, HIPDNN_RNN_WITH_BIAS);
  // HIPDNN_UNIDIRECTIONAL
  EXPECT_EQ(static_cast<int>(parameters.direction_mode_), 0);
  // HIPDNN_DATA_FLOAT
  EXPECT_EQ(static_cast<int>(parameters.data_type_), 0);
  EXPECT_EQ(parameters.data_type_, HIPDNN_DATA_FLOAT);

  EXPECT_EQ(parameters.batch_size_, 64);
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(DefaultParametersTests, GetBidirectionalScaleGetsScale)
{
  {
    DefaultParameters parameters {};

    EXPECT_EQ(parameters.get_bidirectional_scale(), 1);
  }
  {
    DefaultParameters parameters {};

    parameters.direction_mode_ = HIPDNN_BIDIRECTIONAL;

    EXPECT_EQ(parameters.get_bidirectional_scale(), 2);
  }
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(DefaultParametersTests, GetInputTensorSizeGetsSize)
{
  DefaultParameters parameters {};

  EXPECT_EQ(parameters.get_input_tensor_size(), 655360);
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(DefaultParametersTests, GetOutputTensorSizeGetsSize)
{
  DefaultParameters parameters {};

  EXPECT_EQ(parameters.get_output_tensor_size(), 655360);
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(DefaultParametersTests, GetHiddenTensorSizeGetsSize)
{
  DefaultParameters parameters {};

  EXPECT_EQ(parameters.get_hidden_tensor_size(), 65536);
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(DefaultParametersTests, GetTotalMemoryConsumptionGetsSize)
{
  DefaultParameters parameters {};

  EXPECT_EQ(parameters.get_total_memory_consumption<float>(), 12582912);
}

} // namespace RecurrentNeuralNetwork
} // namespace GoogleUnitTests