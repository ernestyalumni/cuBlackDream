#include "RecurrentNeuralNetwork/ManageDescriptor/CellDescriptor.h"
#include "RecurrentNeuralNetwork/ManageDescriptor/HiddenDescriptor.h"
#include "RecurrentNeuralNetwork/ManageDescriptor/InputDescriptor.h"
#include "RecurrentNeuralNetwork/ManageDescriptor/LibraryHandleDropoutRNN.h"
#include "RecurrentNeuralNetwork/ManageDescriptor/OutputDescriptor.h"
#include "RecurrentNeuralNetwork/Modules/Cell.h"
#include "RecurrentNeuralNetwork/Modules/Hidden.h"
#include "RecurrentNeuralNetwork/Modules/Input.h"
#include "RecurrentNeuralNetwork/Modules/Output.h"
#include "RecurrentNeuralNetwork/Operations/forward.h"
#include "RecurrentNeuralNetwork/Parameters.h"
#include "RecurrentNeuralNetwork/SequenceLengthArray.h"
#include "RecurrentNeuralNetwork/WeightSpace.h"
#include "RecurrentNeuralNetwork/WorkAndReserveSpaces.h"
#include "gtest/gtest.h"

#include <hipDNN.h>

using RecurrentNeuralNetwork::DefaultParameters;
using RecurrentNeuralNetwork::HostSequenceLengthArray;
using RecurrentNeuralNetwork::LSTMDefaultParameters;
using RecurrentNeuralNetwork::ManageDescriptor::CellDescriptor;
using RecurrentNeuralNetwork::ManageDescriptor::HiddenDescriptor3Dim;
using RecurrentNeuralNetwork::ManageDescriptor::HiddenDescriptor;
using RecurrentNeuralNetwork::ManageDescriptor::InputDescriptor;
using RecurrentNeuralNetwork::ManageDescriptor::LibraryHandleDropoutRNN;
using RecurrentNeuralNetwork::ManageDescriptor::OutputDescriptor;
using RecurrentNeuralNetwork::Modules::Cell;
using RecurrentNeuralNetwork::Modules::Hidden;
using RecurrentNeuralNetwork::Modules::Input;
using RecurrentNeuralNetwork::Modules::Output;
using RecurrentNeuralNetwork::Operations::forward;
using RecurrentNeuralNetwork::Operations::forward_no_lstm;
using RecurrentNeuralNetwork::SequenceLengthArray;
using RecurrentNeuralNetwork::WeightSpace;
using RecurrentNeuralNetwork::WorkAndReserveSpaces;

namespace GoogleUnitTests
{
namespace RecurrentNeuralNetwork
{
namespace Operations
{

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ForwardTests, Forwards)
{
  DefaultParameters parameters {};
  SequenceLengthArray sequence_length_array {parameters};
  HostSequenceLengthArray host_array {parameters};
  host_array.set_all_to_maximum_sequence_length();
  sequence_length_array.copy_host_input_to_device(host_array);

  InputDescriptor x_descriptor {parameters, sequence_length_array};
  OutputDescriptor y_descriptor {parameters, sequence_length_array};
  LibraryHandleDropoutRNN descriptors {parameters};
  HiddenDescriptor<3> h_descriptor {parameters};
  HiddenDescriptor<3> c_descriptor {parameters};
  h_descriptor.set_strides_by_dimensions();
  c_descriptor.set_strides_by_dimensions();
  h_descriptor.set_descriptor(parameters); 
  c_descriptor.set_descriptor(parameters); 

  WeightSpace weight_space {descriptors};
  WorkAndReserveSpaces spaces {descriptors, x_descriptor};

  Input<float> x {parameters};
  Output<float> y {parameters};
  Hidden<float> hx {parameters};
  Hidden<float> hy {parameters};
  Hidden<float> cx {parameters};
  Hidden<float> cy {parameters};

  const auto result = forward<float, 3>(
    descriptors,
    sequence_length_array,
    x_descriptor,
    x,
    y_descriptor,
    y,
    h_descriptor,
    hx,
    hy,
    c_descriptor,
    cx,
    cy,
    weight_space,
    spaces);

  EXPECT_TRUE(result.is_success());
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ForwardTests, ForwardsOnSequenceMajorPackedLayout)
{
  DefaultParameters parameters {};
  parameters.layout_ = CUDNN_RNN_DATA_LAYOUT_SEQ_MAJOR_PACKED;

  SequenceLengthArray sequence_length_array {parameters};
  HostSequenceLengthArray host_array {parameters};
  host_array.set_all_to_maximum_sequence_length();
  sequence_length_array.copy_host_input_to_device(host_array);

  InputDescriptor x_descriptor {parameters, sequence_length_array};
  OutputDescriptor y_descriptor {parameters, sequence_length_array};

  LibraryHandleDropoutRNN descriptors {parameters};
  HiddenDescriptor<3> h_descriptor {parameters};
  HiddenDescriptor<3> c_descriptor {parameters};
  h_descriptor.set_descriptor(parameters);
  c_descriptor.set_descriptor(parameters);

  WeightSpace weight_space {descriptors};
  WorkAndReserveSpaces spaces {descriptors, x_descriptor};

  Input<float> x {parameters};
  Output<float> y {parameters};
  Hidden<float> hx {parameters};
  Hidden<float> hy {parameters};
  Hidden<float> cx {parameters};
  Hidden<float> cy {parameters};

  const auto result = forward<float, 3>(
    descriptors,
    sequence_length_array,
    x_descriptor,
    x,
    y_descriptor,
    y,
    h_descriptor,
    hx,
    hy,
    c_descriptor,
    cx,
    cy,
    weight_space,
    spaces);

  EXPECT_TRUE(result.is_success());
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ForwardTests, ForwardsOnGRUCellType)
{
  DefaultParameters parameters {};
  parameters.cell_mode_ = HIPDNN_GRU;
  // 512 cells * 6 parameters per cell.
  parameters.input_size_ = 512 * 6;
  parameters.hidden_size_ = 100;
  parameters.projection_size_ = 100;
  parameters.maximum_sequence_length_ = 4;
  parameters.batch_size_ = 10;
  parameters.auxiliary_flags_ = CUDNN_RNN_PADDED_IO_ENABLED;
  parameters.layout_ = CUDNN_RNN_DATA_LAYOUT_BATCH_MAJOR_UNPACKED;

  EXPECT_TRUE(parameters.check_for_valid_parameters());

  SequenceLengthArray sequence_length_array {parameters};
  {
    HostSequenceLengthArray host_array {parameters};
    host_array.set_all_to_maximum_sequence_length();
    sequence_length_array.copy_host_input_to_device(host_array);
  }

  InputDescriptor x_descriptor {parameters, sequence_length_array};
  OutputDescriptor y_descriptor {parameters, sequence_length_array};

  LibraryHandleDropoutRNN descriptors {parameters};
  HiddenDescriptor<3> h_descriptor {parameters};
  HiddenDescriptor<3> c_descriptor {parameters};
  h_descriptor.set_descriptor(parameters);
  c_descriptor.set_descriptor(parameters);

  WeightSpace weight_space {descriptors};
  WorkAndReserveSpaces spaces {descriptors, x_descriptor};

  Input<float> x {parameters};
  Output<float> y {parameters};
  Hidden<float> hx {parameters};
  Hidden<float> hy {parameters};
  Hidden<float> cx {parameters};
  Hidden<float> cy {parameters};

  const auto result = forward<float, 3>(
    descriptors,
    sequence_length_array,
    x_descriptor,
    x,
    y_descriptor,
    y,
    h_descriptor,
    hx,
    hy,
    c_descriptor,
    cx,
    cy,
    weight_space,
    spaces);

  EXPECT_TRUE(result.is_success());
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ForwardTests, ForwardsOnBatchMajorPackedDataLayout)
{
  DefaultParameters parameters {};
  parameters.cell_mode_ = HIPDNN_GRU;
  // 512 cells * 6 parameters per cell.
  parameters.input_size_ = 512 * 6;
  parameters.hidden_size_ = 100;
  parameters.projection_size_ = 100;
  parameters.maximum_sequence_length_ = 4;
  parameters.batch_size_ = 10;
  parameters.auxiliary_flags_ = CUDNN_RNN_PADDED_IO_ENABLED;
  parameters.layout_ = CUDNN_RNN_DATA_LAYOUT_BATCH_MAJOR_UNPACKED;

  EXPECT_TRUE(parameters.check_for_valid_parameters());

  SequenceLengthArray sequence_length_array {parameters};
  HostSequenceLengthArray host_array {parameters};
  host_array.set_all_to_maximum_sequence_length();
  sequence_length_array.copy_host_input_to_device(host_array);

  InputDescriptor x_descriptor {parameters, sequence_length_array};
  OutputDescriptor y_descriptor {parameters, sequence_length_array};

  LibraryHandleDropoutRNN descriptors {parameters};
  HiddenDescriptor<3> h_descriptor {parameters};
  HiddenDescriptor<3> c_descriptor {parameters};
  h_descriptor.set_descriptor(parameters);
  c_descriptor.set_descriptor(parameters);

  WeightSpace weight_space {descriptors};
  WorkAndReserveSpaces spaces {descriptors, x_descriptor};

  Input<float> x {parameters};
  Output<float> y {parameters};
  Hidden<float> hx {parameters};
  Hidden<float> hy {parameters};
  Hidden<float> cx {parameters};
  Hidden<float> cy {parameters};

  const auto result = forward<float, 3>(
    descriptors,
    sequence_length_array,
    x_descriptor,
    x,
    y_descriptor,
    y,
    h_descriptor,
    hx,
    hy,
    c_descriptor,
    cx,
    cy,
    weight_space,
    spaces);

  EXPECT_TRUE(result.is_success());
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ForwardTests, ForwardsOnLSTM)
{
  LSTMDefaultParameters parameters {
    29,
    50,
    10,
    1,
    28,
    100};

  EXPECT_TRUE(parameters.check_for_valid_parameters());

  SequenceLengthArray sequence_length_array {parameters};
  {
    HostSequenceLengthArray host_array {parameters};
    host_array.set_all_to_maximum_sequence_length();
    sequence_length_array.copy_host_input_to_device(host_array);
  }

  InputDescriptor x_descriptor {parameters, sequence_length_array};
  OutputDescriptor y_descriptor {parameters, sequence_length_array};

  LibraryHandleDropoutRNN descriptors {parameters};
  HiddenDescriptor<3> h_descriptor {parameters};
  CellDescriptor<3> c_descriptor {parameters};
  h_descriptor.set_descriptor(parameters);
  c_descriptor.set_descriptor(parameters);

  WeightSpace weight_space {descriptors};
  WorkAndReserveSpaces spaces {descriptors, x_descriptor};

  Input<float> x {parameters};
  Output<float> y {parameters};
  Hidden<float> hx {parameters};
  Hidden<float> hy {parameters};
  Cell<float> cx {parameters};
  Cell<float> cy {parameters};

  const auto result = forward<float, 3>(
    descriptors,
    sequence_length_array,
    x_descriptor,
    x,
    y_descriptor,
    y,
    h_descriptor,
    hx,
    hy,
    c_descriptor,
    cx,
    cy,
    weight_space,
    spaces);

  EXPECT_TRUE(result.is_success());
}

//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
TEST(ForwardTests, ForwardsOnLSTMAndNoSequenceLengths)
{
  LSTMDefaultParameters parameters {
    29,
    50,
    10,
    1,
    28,
    100};

  EXPECT_TRUE(parameters.check_for_valid_parameters());

  SequenceLengthArray sequence_length_array {parameters};
  {
    HostSequenceLengthArray host_array {parameters};
    host_array.set_all_to_maximum_sequence_length();
    sequence_length_array.copy_host_input_to_device(host_array);
  }

  InputDescriptor x_descriptor {parameters, sequence_length_array};
  OutputDescriptor y_descriptor {parameters, sequence_length_array};

  LibraryHandleDropoutRNN descriptors {parameters};
  HiddenDescriptor<3> h_descriptor {parameters};
  CellDescriptor<3> c_descriptor {parameters};
  h_descriptor.set_descriptor(parameters);
  c_descriptor.set_descriptor(parameters);

  WeightSpace weight_space {descriptors};
  WorkAndReserveSpaces spaces {descriptors, x_descriptor};

  Input<float> x {parameters};
  Output<float> y {parameters};
  Hidden<float> hx {parameters};
  Hidden<float> hy {parameters};
  Cell<float> cx {parameters};
  Cell<float> cy {parameters};

  const auto result = forward<float, 3>(
    descriptors,
    x_descriptor,
    x,
    y_descriptor,
    y,
    h_descriptor,
    hx,
    hy,
    c_descriptor,
    cx,
    cy,
    weight_space,
    spaces);

  EXPECT_TRUE(result.is_success());
}

} // namespace Operations
} // namespace RecurrentNeuralNetwork
} // namespace GoogleUnitTests